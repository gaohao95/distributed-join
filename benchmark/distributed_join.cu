#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cstdint>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <memory>
#include <stdexcept>
#include <string>
#include <tuple>
#include <utility>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <mpi.h>

#include <cudf/table/table.hpp>
#include <cudf/types.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

#include "../src/communicator.h"
#include "../src/distributed_join.cuh"
#include "../src/error.cuh"
#include "../src/generate_table.cuh"
#include "../src/registered_memory_resource.hpp"
#include "../src/topology.cuh"

static std::string key_type     = "int64_t";
static std::string payload_type = "int64_t";

static cudf::size_type BUILD_TABLE_NROWS_EACH_RANK = 100'000'000;
static cudf::size_type PROBE_TABLE_NROWS_EACH_RANK = 100'000'000;
static double SELECTIVITY                          = 0.3;
static bool IS_BUILD_TABLE_KEY_UNIQUE              = true;
static int OVER_DECOMPOSITION_FACTOR               = 1;
static std::string COMMUNICATOR_NAME               = "UCX";
static std::string REGISTRATION_METHOD             = "preregistered";
static int64_t COMMUNICATOR_BUFFER_SIZE            = 1'600'000'000LL;

void parse_command_line_arguments(int argc, char *argv[])
{
  for (int iarg = 0; iarg < argc; iarg++) {
    if (!strcmp(argv[iarg], "--key-type")) { key_type = argv[iarg + 1]; }

    if (!strcmp(argv[iarg], "--payload-type")) { payload_type = argv[iarg + 1]; }

    if (!strcmp(argv[iarg], "--build-table-nrows")) {
      BUILD_TABLE_NROWS_EACH_RANK = atoi(argv[iarg + 1]);
    }

    if (!strcmp(argv[iarg], "--probe-table-nrows")) {
      PROBE_TABLE_NROWS_EACH_RANK = atoi(argv[iarg + 1]);
    }

    if (!strcmp(argv[iarg], "--selectivity")) { SELECTIVITY = atof(argv[iarg + 1]); }

    if (!strcmp(argv[iarg], "--duplicate-build-keys")) { IS_BUILD_TABLE_KEY_UNIQUE = false; }

    if (!strcmp(argv[iarg], "--over-decomposition-factor")) {
      OVER_DECOMPOSITION_FACTOR = atoi(argv[iarg + 1]);
    }

    if (!strcmp(argv[iarg], "--communicator")) { COMMUNICATOR_NAME = argv[iarg + 1]; }

    if (!strcmp(argv[iarg], "--registration-method")) { REGISTRATION_METHOD = argv[iarg + 1]; }
  }
}

void report_configuration()
{
  MPI_CALL(MPI_Barrier(MPI_COMM_WORLD));

  int mpi_rank;
  int mpi_size;
  MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank));
  MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &mpi_size));
  if (mpi_rank != 0) return;

  std::cout << "========== Parameters ==========" << std::endl;
  std::cout << std::boolalpha;
  std::cout << "Key type: " << key_type << std::endl;
  std::cout << "Payload type: " << payload_type << std::endl;
  std::cout << "Number of rows in the build table: "
            << static_cast<uint64_t>(BUILD_TABLE_NROWS_EACH_RANK) * mpi_size / 1e6 << " million"
            << std::endl;
  std::cout << "Number of rows in the probe table: "
            << static_cast<uint64_t>(PROBE_TABLE_NROWS_EACH_RANK) * mpi_size / 1e6 << " million"
            << std::endl;
  std::cout << "Selectivity: " << SELECTIVITY << std::endl;
  std::cout << "Keys in build table are unique: " << IS_BUILD_TABLE_KEY_UNIQUE << std::endl;
  std::cout << "Over-decomposition factor: " << OVER_DECOMPOSITION_FACTOR << std::endl;
  std::cout << "Communicator: " << COMMUNICATOR_NAME << std::endl;
  if (COMMUNICATOR_NAME == "UCX")
    std::cout << "Registration method: " << REGISTRATION_METHOD << std::endl;
  std::cout << "================================" << std::endl;
}

int main(int argc, char *argv[])
{
  /* Initialize topology */

  setup_topology(argc, argv);

  /* Parse command line arguments */

  parse_command_line_arguments(argc, argv);
  report_configuration();

  cudf::size_type RAND_MAX_VAL =
    std::max(BUILD_TABLE_NROWS_EACH_RANK, PROBE_TABLE_NROWS_EACH_RANK) * 2;

  /* Initialize communicator and memory pool */

  int mpi_rank;
  int mpi_size;
  MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank));
  MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &mpi_size));

  Communicator *communicator{nullptr};
  // `registered_mr` holds reference to the registered memory resource, and *nullptr* if registered
  // memory resource is not used.
  registered_memory_resource *registered_mr{nullptr};
  // pool_mr need to live on heap because for registered memory resources, the memory pool needs
  // to deallocated before UCX cleanup, which can be achieved by calling the destructor of
  // `poll_mr`.
  rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource> *pool_mr{nullptr};

  setup_memory_pool_and_communicator(communicator,
                                     registered_mr,
                                     pool_mr,
                                     COMMUNICATOR_NAME,
                                     REGISTRATION_METHOD,
                                     COMMUNICATOR_BUFFER_SIZE);

  /* Generate build table and probe table on each rank */

  std::unique_ptr<cudf::table> left;
  std::unique_ptr<cudf::table> right;

#define generate_tables(KEY_T, PAYLOAD_T)                                        \
  {                                                                              \
    std::tie(left, right) =                                                      \
      generate_tables_distributed<KEY_T, PAYLOAD_T>(BUILD_TABLE_NROWS_EACH_RANK, \
                                                    PROBE_TABLE_NROWS_EACH_RANK, \
                                                    SELECTIVITY,                 \
                                                    RAND_MAX_VAL,                \
                                                    IS_BUILD_TABLE_KEY_UNIQUE,   \
                                                    communicator);               \
  }

#define generate_tables_key_type(KEY_T)                 \
  {                                                     \
    if (payload_type == "int64_t") {                    \
      generate_tables(KEY_T, int64_t)                   \
    } else if (payload_type == "int32_t") {             \
      generate_tables(KEY_T, int32_t)                   \
    } else {                                            \
      throw std::runtime_error("Unknown payload type"); \
    }                                                   \
  }

  if (key_type == "int64_t") {
    generate_tables_key_type(int64_t)
  } else if (key_type == "int32_t") {
    generate_tables_key_type(int32_t)
  } else {
    throw std::runtime_error("Unknown key type");
  }

  /* Distributed join */

  CUDA_RT_CALL(hipDeviceSynchronize());

  MPI_Barrier(MPI_COMM_WORLD);
  hipProfilerStart();
  double start = MPI_Wtime();

  std::unique_ptr<cudf::table> join_result =
    distributed_inner_join(left->view(),
                           right->view(),
                           {0},
                           {0},
                           {std::pair<cudf::size_type, cudf::size_type>(0, 0)},
                           communicator,
                           OVER_DECOMPOSITION_FACTOR);

  MPI_Barrier(MPI_COMM_WORLD);
  double stop = MPI_Wtime();
  hipProfilerStop();

  if (mpi_rank == 0) { std::cout << "Elasped time (s) " << stop - start << std::endl; }

  /* Cleanup */
  left.reset();
  right.reset();
  join_result.reset();
  CUDA_RT_CALL(hipDeviceSynchronize());

  destroy_memory_pool_and_communicator(
    communicator, registered_mr, pool_mr, COMMUNICATOR_NAME, REGISTRATION_METHOD);

  MPI_CALL(MPI_Finalize());

  return 0;
}
